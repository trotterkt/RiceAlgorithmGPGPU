/*
 ============================================================================
 Name        : RiceAlgorithmGPGPU.cu
 Author      : Keir Trotter
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <iostream>
#include <FileBasedImagePersistence.h>
#include <Sensor.h>
#include <CudaHelper.h>
#include <RiceAlgorithmKernels.h>

using namespace std;
using namespace RiceAlgorithm;



/**
 * Host function that copies the data and launches the work on GPU
 */
float *gpuReciprocal(float *data, unsigned size)
{
	float *rc = new float[size];
	float *gpuData;

	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuData, sizeof(float)*size));
	CUDA_CHECK_RETURN(hipMemcpy(gpuData, data, sizeof(float)*size, hipMemcpyHostToDevice));
	
	static const int BLOCK_SIZE = 256;
	const int blockCount = (size+BLOCK_SIZE-1)/BLOCK_SIZE;
	reciprocalKernel<<<blockCount, BLOCK_SIZE>>> (gpuData, size);

	CUDA_CHECK_RETURN(hipMemcpy(rc, gpuData, sizeof(float)*size, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(gpuData));
	return rc;
}

float *cpuReciprocal(float *data, unsigned size)
{
	float *rc = new float[size];
	for (unsigned cnt = 0; cnt < size; ++cnt) rc[cnt] = 1.0/data[cnt];
	return rc;
}


void initialize(float *data, unsigned size)
{
	for (unsigned i = 0; i < size; ++i)
		data[i] = .5*(i+1);
}

int main(void)
{
	//=====================================================

    cout.precision(4);

    cout << "Compressing Landsat_agriculture-u16be-6x1024x1024..." << endl;

    FileBasedImagePersistence image("Landsat_agriculture-u16be-6x1024x1024", Rows, Columns, Bands);


    // This data has not been pre-processed. Need to decide if the pre-processor will be
    // placed on the GPGPU -- if so, needs to be un-associated from the Sensor type

//    ushort* hostImagePtr = image.getSampleData(1);
//
//
//    ushort *gpuRawImageData;
//
//	const int NumberOfSamples(Rows * Columns * Bands);
//
//
//	CUDA_CHECK_RETURN(hipMalloc((void **)&gpuRawImageData, sizeof(ushort)*NumberOfSamples));
//	CUDA_CHECK_RETURN(hipMemcpy(gpuRawImageData, hostImagePtr, sizeof(ushort)*NumberOfSamples, hipMemcpyHostToDevice));



    // Construct my LandSat sensor, which performs the compression of the supplied
    // raw image data per the Rice algorithm
	Sensor landsat(&image, Rows, Columns, Bands);

    // Initiate the Rice algorithm compression
	landsat.process();

    //=====================================================

//	static const int WORK_SIZE = 65530;
//	float *data = new float[WORK_SIZE];
//
//	initialize (data, WORK_SIZE);
//
//	float *recCpu = cpuReciprocal(data, WORK_SIZE);
//	float *recGpu = gpuReciprocal(data, WORK_SIZE);
//	float cpuSum = std::accumulate (recCpu, recCpu+WORK_SIZE, 0.0);
//	float gpuSum = std::accumulate (recGpu, recGpu+WORK_SIZE, 0.0);
//
//	/* Verify the results */
//	std::cout<<"gpuSum = "<<gpuSum<< " cpuSum = " <<cpuSum<<std::endl;
//
//	/* Free memory */
//	delete[] data;
//	delete[] recCpu;
//	delete[] recGpu;

	hipDeviceReset();

	return 0;
}


